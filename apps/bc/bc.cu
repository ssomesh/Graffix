#include "hip/hip_runtime.h"
#include "bc.h"

void bc_exact (Graph& G, double* nodeBC) {
  
  // declare the auxillary data structures for the bc computation
  
  unsigned n = G.nnodes;
  unsigned * d_sigma;
  int hops_from_source, * d_level, * d_hops_from_source;
  double * d_delta, * d_nodeBC;
  bool takeNextIter, * d_takeNextIter; // to decide whether to go for the next iteration or not
  gpuErrchk(hipMalloc(&d_sigma, sizeof(unsigned) * n));
  gpuErrchk(hipMalloc(&d_level, sizeof(int) * n));
  gpuErrchk(hipMalloc(&d_delta, sizeof(double) * n));
  gpuErrchk(hipMalloc(&d_nodeBC, sizeof(double) * n));

  gpuErrchk(hipMalloc(&d_hops_from_source, sizeof(int)));
  gpuErrchk(hipMalloc(&d_takeNextIter, sizeof(bool)));
  
  gpuErrchk(hipMemset(d_nodeBC, 0, sizeof(double)*n)); // initializing node bc

  
  unsigned blockSize = 512;
  unsigned gridSize = (n + blockSize - 1) / blockSize; // ceil(n/blockSize)
  
  for(unsigned s=0; s<n; ++s) {  // outer loop of Brandes' Algorithm
     hops_from_source = 0; // keeps track of the number of hops from source in the current iteration. 
     
     initialize<<<gridSize, blockSize>>>(d_sigma, d_delta, d_level, d_hops_from_source, s, n);
#ifdef DEBUG
     hipDeviceSynchronize();
     hipError_t errCode = hipPeekAtLastError();
     if (errCode != hipSuccess) {
       fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n",
       errCode, hipGetErrorString(errCode));
    }
#endif
    
    // forward pass 
    do {
      hipMemset(d_takeNextIter,false,sizeof(bool));
      bc_forward_pass<<<gridSize, blockSize>>>(G, d_sigma, d_level, d_hops_from_source, n, d_takeNextIter);
      hipDeviceSynchronize();
#ifdef DEBUG
     hipError_t errCode = hipPeekAtLastError();
     if (errCode != hipSuccess) {
       fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n",
       errCode, hipGetErrorString(errCode));
    }
#endif

      ++hops_from_source; // updating the level to process in the next iteration
//      gpuErrchk(hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(hops_from_source), hipMemcpyHostToDevice));
      incHop<<<1,1>>>(d_hops_from_source);

      gpuErrchk(hipMemcpy(&takeNextIter, d_takeNextIter, sizeof(bool), hipMemcpyDeviceToHost));
    }while(takeNextIter);
    
    
  // backward pass
  
  --hops_from_source;
  gpuErrchk(hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(hops_from_source), hipMemcpyHostToDevice));
  while(hops_from_source > 1) {
    bc_backward_pass<<<gridSize,blockSize>>>(G, d_sigma, d_delta, d_nodeBC, d_level, d_hops_from_source, n);
#ifdef DEBUG
     hipError_t errCode = hipPeekAtLastError();
     if (errCode != hipSuccess) {
       fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n",
       errCode, hipGetErrorString(errCode));
    }
#endif
    --hops_from_source;
    gpuErrchk(hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(hops_from_source), hipMemcpyHostToDevice));
  }

  accumulate_bc<<<gridSize, blockSize>>>(d_delta, d_nodeBC, d_level, s, n);
#ifdef DEBUG
     errCode = hipPeekAtLastError();
     if (errCode != hipSuccess) {
       fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n",
       errCode, hipGetErrorString(errCode));
    }
#endif
  hipDeviceSynchronize();



  } // end of outerloop of brandes' algorithm
   


    gpuErrchk(hipMemcpy(nodeBC, d_nodeBC, sizeof(double) * n, hipMemcpyDeviceToHost));

}



__global__ void initialize(unsigned* d_sigma, double* d_delta, int* d_level, int* d_hops_from_source, unsigned s, unsigned n) {
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= n) return;
    d_level[tid] = -1;
    d_delta[tid] = 0.0;
    d_sigma[tid] = 0;
    
    if(tid == s) { // for the source
      d_level[tid] = 0;
      d_sigma[tid] = 1;
      *d_hops_from_source = 0;
    }
  }

  __global__ void bc_forward_pass(Graph G, unsigned* d_sigma, int* d_level, int* d_hops_from_source, unsigned n, bool* d_takeNextIter) {
    unsigned u = blockIdx.x * blockDim.x + threadIdx.x;
    if(u >= n) return;

    // only processing the nodes at level '*d_hops_from_source' -- a level synchronous processing, though not work efficient
    if(d_level[u] == *d_hops_from_source) {  
       unsigned end = G.d_offset[u+1];
       for(unsigned i = G.d_offset[u]; i < end; ++i) { // going over the neighbors of u
          unsigned v = G.d_edges[i];
          if(d_level[v] == -1) {  // v is seen for the first time
            d_level[v] = *d_hops_from_source + 1; // no atomics required since this is benign data race due to level synchronous implementation
            *d_takeNextIter = true;
          }
          if(d_level[v] == *d_hops_from_source + 1) { // 'v' is indeed the neighbor of u
            atomicAdd(&d_sigma[v], d_sigma[u]);
          }
       }
    }
  }

  __global__ void bc_backward_pass(Graph G, unsigned* d_sigma, double* d_delta, double* d_nodeBC, int* d_level, int* d_hops_from_source, unsigned n) {
      
    unsigned u = blockIdx.x * blockDim.x + threadIdx.x;
    if(u >= n) return;

    if(d_level[u] == *d_hops_from_source - 1) { // backward traversal of DAG, one level at a time 
      
       unsigned end = G.d_offset[u+1];
       double sum = 0.0;
       for(unsigned i = G.d_offset[u]; i < end; ++i) { // going over the neighbors of u for which it is the predecessor in the DAG
          unsigned v = G.d_edges[i];
          if(d_level[v] == *d_hops_from_source) {
            sum += (1.0 * d_sigma[u]) / d_sigma[v] * (1.0 + d_delta[v]);
          }
       }

       d_delta[u] += sum;

    }

  }

  __global__ void accumulate_bc(double * d_delta, double* d_nodeBC, int* d_level, unsigned s, unsigned n) {
    
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= n || tid == s || d_level[tid] == -1) return;

    d_nodeBC[tid] += d_delta[tid]/2.0;

  }

__global__ void incHop(int* d_hops_from_source) {
    *d_hops_from_source = *d_hops_from_source + 1;
  }
