#include "hip/hip_runtime.h"
#include <iostream>
#include "graph.h"
#include "utils.h"
#include "timer.h"
#include <stdio.h>
#include <cstring>
#include <cstdlib>

__global__ void dinit(unsigned *mstwt, Graph graph, ComponentSpace cs, foru *eleminwts, foru *minwtcomponent, unsigned *partners, unsigned *phores, bool *processinnextiteration, unsigned *goaheadnodeofcomponent, unsigned inpid) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if (inpid < graph.nnodes) id = inpid;

	if (id < graph.nnodes) {
		eleminwts[id] = MYINFINITY;
		minwtcomponent[id] = MYINFINITY;	
		goaheadnodeofcomponent[id] = graph.nnodes;
		phores[id] = 0;
		partners[id] = id;
		processinnextiteration[id] = false;
	}
}
__global__ void dfindelemin(unsigned *mstwt, Graph graph, ComponentSpace cs, foru *eleminwts, foru *minwtcomponent, unsigned *partners, unsigned *phore, bool *processinnextiteration, unsigned *goaheadnodeofcomponent, unsigned inpid) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if (inpid < graph.nnodes) id = inpid;

	if (id < graph.nnodes) {
		// if I have a cross-component edge,
		// 	find my minimum wt cross-component edge,
		//	inform my boss about this edge e (atomicMin).
		unsigned src = id;
		unsigned srcboss = cs.find(src);
		unsigned dstboss = graph.nnodes;
		foru minwt = MYINFINITY;
		unsigned degree = graph.getOutDegree(src);
		for (unsigned ii = 0; ii < degree; ++ii) {
			foru wt = graph.getWeight(src, ii);
			if (wt < minwt) {
				unsigned dst = graph.getDestination(src, ii);
				unsigned tempdstboss = cs.find(dst);
				if (srcboss != tempdstboss) {	// cross-component edge.
					minwt = wt;
					dstboss = tempdstboss;
				}
			}
		}
		dprintf("\tminwt[%d] = %d\n", id, minwt);
		eleminwts[id] = minwt;
		partners[id] = dstboss;

		if (minwt < minwtcomponent[srcboss] && srcboss != dstboss) {
			// inform boss.
			foru oldminwt = atomicMin(&minwtcomponent[srcboss], minwt);
		}
	}
}

__global__ void dfindelemin2(unsigned *mstwt, Graph graph, ComponentSpace cs, foru *eleminwts, foru *minwtcomponent, unsigned *partners, unsigned *phore, bool *processinnextiteration, unsigned *goaheadnodeofcomponent, unsigned inpid) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (id < graph.nnodes) {
		unsigned src = id;
		unsigned srcboss = cs.find(src);

		if(eleminwts[id] == minwtcomponent[srcboss] && srcboss != partners[id] && partners[id] != graph.nnodes)
		  {
		    unsigned degree = graph.getOutDegree(src);
		    for (unsigned ii = 0; ii < degree; ++ii) {
		      foru wt = graph.getWeight(src, ii);
		      if (wt == eleminwts[id]) {
			unsigned dst = graph.getDestination(src, ii);
			unsigned tempdstboss = cs.find(dst);
			if (tempdstboss == partners[id]) {	// cross-component edge.
			  //atomicMin(&goaheadnodeofcomponent[srcboss], id);
			  
			  if(atomicCAS(&goaheadnodeofcomponent[srcboss], graph.nnodes, id) == graph.nnodes)
			    {
			    }
			}
		      }
		    }
		  }
	}
}



__global__ void verify_min_elem(unsigned *mstwt, Graph graph, ComponentSpace cs, foru *eleminwts, foru *minwtcomponent, unsigned *partners, unsigned *phore, bool *processinnextiteration, unsigned *goaheadnodeofcomponent, unsigned inpid) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if (inpid < graph.nnodes) id = inpid;

	if (id < graph.nnodes) {
	  if(cs.isBoss(id))
	    {
	      if(goaheadnodeofcomponent[id] == graph.nnodes)
		{
		  return;
		}

	      unsigned minwt_node = goaheadnodeofcomponent[id];

	      unsigned degree = graph.getOutDegree(minwt_node);
	      foru minwt = minwtcomponent[id];

	      if(minwt == MYINFINITY)
		return;
		
	      bool minwt_found = false;
	      for (unsigned ii = 0; ii < degree; ++ii) {
		foru wt = graph.getWeight(minwt_node, ii);

		if (wt == minwt) {
		  minwt_found = true;
		  unsigned dst = graph.getDestination(minwt_node, ii);
		  unsigned tempdstboss = cs.find(dst);
		  if(tempdstboss == partners[minwt_node] && tempdstboss != id)
		    {
		      processinnextiteration[minwt_node] = true;
		      return;
		    }
		}
	      }

	      printf("component %d is wrong %d\n", id, minwt_found);
	    }
	}
}

__global__ void elim_dups(unsigned *mstwt, Graph graph, ComponentSpace cs, foru *eleminwts, foru *minwtcomponent, unsigned *partners, unsigned *phore, bool *processinnextiteration, unsigned *goaheadnodeofcomponent, unsigned inpid) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if (inpid < graph.nnodes) id = inpid;

	if (id < graph.nnodes) {
	  if(processinnextiteration[id])
	    {
	      unsigned srcc = cs.find(id);
	      unsigned dstc = partners[id];
	      
	      if(minwtcomponent[dstc] == eleminwts[id])
		{
		  if(id < goaheadnodeofcomponent[dstc])
		    {
		      processinnextiteration[id] = false;
		    }
		}
	    }
	}
}

__global__ void dfindcompmin(unsigned *mstwt, Graph graph, ComponentSpace cs, foru *eleminwts, foru *minwtcomponent, unsigned *partners, unsigned *phores, bool *processinnextiteration, unsigned *goaheadnodeofcomponent, unsigned inpid) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if (inpid < graph.nnodes) id = inpid;

	if (id < graph.nnodes) {
	  if(partners[id] == graph.nnodes)
	    return;

	  unsigned srcboss = cs.find(id);
	  unsigned dstboss = cs.find(partners[id]);
	  if (id != partners[id] && srcboss != dstboss && eleminwts[id] != MYINFINITY && minwtcomponent[srcboss] == eleminwts[id] && dstboss != id && goaheadnodeofcomponent[srcboss] == id) {	// my edge is min outgoing-component edge.
	    if(!processinnextiteration[id]);
	  }
	  else
	    {
	      if(processinnextiteration[id]);
	    }
	}
}

__global__ void dfindcompmintwo(unsigned *mstwt, Graph graph, ComponentSpace csw, foru *eleminwts, foru *minwtcomponent, unsigned *partners, unsigned *phores, bool *processinnextiteration, unsigned *goaheadnodeofcomponent, unsigned inpid, GlobalBarrier gb, bool *repeat, unsigned *count) {
	unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned id, nthreads = blockDim.x * gridDim.x;
	if (inpid < graph.nnodes) id = inpid;

	unsigned up = (graph.nnodes + nthreads - 1) / nthreads * nthreads;
	unsigned srcboss, dstboss;


	for(id = tid; id < up; id += nthreads) {
	  if(id < graph.nnodes && processinnextiteration[id])
	    {
	      srcboss = csw.find(id);
	      dstboss = csw.find(partners[id]);
	    }
	  
	  gb.Sync();
	  	  
	  if (id < graph.nnodes && processinnextiteration[id] && srcboss != dstboss) {
	    dprintf("trying unify id=%d (%d -> %d)\n", id, srcboss, dstboss);

	    if (csw.unify(srcboss, dstboss)) {
	      atomicAdd(mstwt, eleminwts[id]);
	      atomicAdd(count, 1);
	      dprintf("u %d -> %d (%d)\n", srcboss, dstboss, eleminwts[id]);
	      processinnextiteration[id] = false;
	      eleminwts[id] = MYINFINITY;	// mark end of processing to avoid getting repeated.
	    }
	    else {
	      *repeat = true;
	    }

	    dprintf("\tcomp[%d] = %d.\n", srcboss, csw.find(srcboss));
	  }

	  gb.Sync(); 
	}
}

int main(int argc, char *argv[]) {
  unsigned *mstwt, hmstwt = 0;
  int iteration = 0;
  Graph hgraph, graph;

  unsigned *partners, *phores;
  foru *eleminwts, *minwtcomponent;
  bool *processinnextiteration;
  unsigned *goaheadnodeofcomponent;

  double starttime, endtime;
  GlobalBarrierLifetime gb;
  const size_t compmintwo_res = maximum_residency(dfindcompmintwo, 384, 0);
  gb.Setup(nSM * compmintwo_res);

  if (argc != 2) {
    printf("Usage: %s <graph>\n", argv[0]);
    exit(1);
  }

  hgraph.read(argv[1]);
  hgraph.cudaCopy(graph);

  // preprocessing the graph to make it amenable for coalescing
  renumber_replicate(hgraph); // this will modify the graph
  if (hipMalloc((void **)&mstwt, sizeof(unsigned)) != hipSuccess) CudaTest("allocating mstwt failed");
  CUDA_SAFE_CALL(hipMemcpy(mstwt, &hmstwt, sizeof(hmstwt), hipMemcpyHostToDevice));	// mstwt = 0.

  if (hipMalloc((void **)&eleminwts, graph.nnodes * sizeof(foru)) != hipSuccess) CudaTest("allocating eleminwts failed");
  if (hipMalloc((void **)&minwtcomponent, graph.nnodes * sizeof(foru)) != hipSuccess) CudaTest("allocating minwtcomponent failed");
  if (hipMalloc((void **)&partners, graph.nnodes * sizeof(unsigned)) != hipSuccess) CudaTest("allocating partners failed");
  if (hipMalloc((void **)&phores, graph.nnodes * sizeof(unsigned)) != hipSuccess) CudaTest("allocating phores failed");
  if (hipMalloc((void **)&processinnextiteration, graph.nnodes * sizeof(bool)) != hipSuccess) CudaTest("allocating processinnextiteration failed");
  if (hipMalloc((void **)&goaheadnodeofcomponent, graph.nnodes * sizeof(unsigned)) != hipSuccess) CudaTest("allocating goaheadnodeofcomponent failed");

  

  unsigned prevncomponents, currncomponents = graph.nnodes;

  bool repeat = false, *grepeat;
  CUDA_SAFE_CALL(hipMalloc(&grepeat, sizeof(bool) * 1));
  CUDA_SAFE_CALL(hipMemcpy(grepeat, &repeat, sizeof(bool) * 1, hipMemcpyHostToDevice));

  unsigned edgecount = 0, *gedgecount;
  CUDA_SAFE_CALL(hipMalloc(&gedgecount, sizeof(unsigned) * 1));
  CUDA_SAFE_CALL(hipMemcpy(gedgecount, &edgecount, sizeof(unsigned) * 1, hipMemcpyHostToDevice));

  printf("finding mst.\n");
  starttime = rtclock();

  do {
    ++iteration;
    prevncomponents = currncomponents;
    dinit 		<<<num_blocks, block_size>>> (mstwt, graph, cs, eleminwts, minwtcomponent, partners, phores, processinnextiteration, goaheadnodeofcomponent, graph.nnodes);
    CudaTest("dinit failed");
    dfindelemin 	<<<num_blocks, block_size>>> (mstwt, graph, cs, eleminwts, minwtcomponent, partners, phores, processinnextiteration, goaheadnodeofcomponent, graph.nnodes);
    dfindelemin2 	<<<num_blocks, block_size>>> (mstwt, graph, cs, eleminwts, minwtcomponent, partners, phores, processinnextiteration, goaheadnodeofcomponent, graph.nnodes);
    verify_min_elem 	<<<num_blocks, block_size>>> (mstwt, graph, cs, eleminwts, minwtcomponent, partners, phores, processinnextiteration, goaheadnodeofcomponent, graph.nnodes);
    CudaTest("dfindelemin failed");
    if(debug) print_comp_mins(cs, graph, minwtcomponent, goaheadnodeofcomponent, partners, processinnextiteration);


    do {
      repeat = false;

      CUDA_SAFE_CALL(hipMemcpy(grepeat, &repeat, sizeof(bool) * 1, hipMemcpyHostToDevice));
      dfindcompmintwo <<<nSM * compmintwo_res, 384>>> (mstwt, graph, cs, eleminwts, minwtcomponent, partners, phores, processinnextiteration, goaheadnodeofcomponent, graph.nnodes, gb, grepeat, gedgecount);
      CudaTest("dfindcompmintwo failed");
		  
      CUDA_SAFE_CALL(hipMemcpy(&repeat, grepeat, sizeof(bool) * 1, hipMemcpyDeviceToHost));
    } while (repeat); 

    currncomponents = cs.numberOfComponentsHost();
    CUDA_SAFE_CALL(hipMemcpy(&hmstwt, mstwt, sizeof(hmstwt), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(&edgecount, gedgecount, sizeof(unsigned) * 1, hipMemcpyDeviceToHost));
      printf("\titeration %d, number of components = %d , mstwt = %u mstedges = %u\n", iteration, currncomponents, hmstwt, edgecount);

      edgecount = 0; // reinitializing for the next iteration
    CUDA_SAFE_CALL(hipMemcpy(gedgecount,&edgecount, sizeof(unsigned) * 1, hipMemcpyHostToDevice));

      
  } while (currncomponents != prevncomponents);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  endtime = rtclock();
	
  printf("\tmstwt = %u, iterations = %d.\n", hmstwt, iteration);
  printf("\t%s result: weight: %u, components: %u, edges: %u\n", argv[1], hmstwt, currncomponents, edgecount);
  printf("\truntime [mst] = %f ms.\n", 1000 * (endtime - starttime));


  return 0;
}

/*renumber and replicate the nodes */
void renumber_replicate(Graph& G) { 
  /* Step-1: renumber the nodes */

  // store the nodes' degrees in an array and sort the array in descending order
  int * h_nodeDegree = (int*) malloc(G.h_nnodes*sizeof(int));
  int * h_nodeDegree_aux = (int*) malloc(G.h_nnodes*sizeof(int)); // this is for the merge sort
  int * d_nodeDegree;
  uint64_t * h_nodeId = (uint64_t*) malloc(G.h_nnodes*sizeof(uint64_t));
  uint64_t * h_nodeId_aux = (uint64_t*) malloc(G.h_nnodes*sizeof(uint64_t));
  uint64_t * d_nodeId;
  gpuErrchk(hipMalloc(&d_nodeId,G.h_nnodes*sizeof(uint64_t))); 
  gpuErrchk(hipMalloc(&d_nodeDegree,G.h_nnodes*sizeof(int))); 
  unsigned blockSize = 256;
  unsigned numBlocks = (G.h_nnodes+blockSize-1)/blockSize;

  CPUTimer cputimer;
  cputimer.Start();

  populateDegree<<<numBlocks, blockSize>>>(G, d_nodeDegree,d_nodeId,G.h_nnodes);
  gpuErrchk(hipMemcpy(h_nodeDegree, d_nodeDegree, G.h_nnodes*sizeof(int), hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(h_nodeId, d_nodeId, G.h_nnodes*sizeof(uint64_t), hipMemcpyDeviceToHost));  

  gpuErrchk(hipFree(d_nodeDegree));
  gpuErrchk(hipFree(d_nodeId));
  

  // sort the nodes in descending order and maintain another array to store the corresponding node id  
   Merge_Sort_Par(h_nodeDegree, h_nodeDegree_aux, h_nodeId, h_nodeId_aux, G.h_nnodes);

  cputimer.Stop();

  free(h_nodeDegree_aux);
  free(h_nodeId_aux);

  std::cout << "Time elapsed = " << cputimer.Elapsed() << " second" << std::endl;

  
#if 1

  int *h_level = (int*) malloc(G.h_nnodes*sizeof(int));
  int * d_level;
  gpuErrchk(hipMalloc(&d_level,G.h_nnodes*sizeof(int))); 

  cputimer.Start();

  levelInit<<<numBlocks,blockSize>>>(d_level,G.h_nnodes);

  gpuErrchk(hipMemcpy(h_level, d_level, G.h_nnodes*sizeof(int), hipMemcpyDeviceToHost));  // initializing h_level for the first iteration

  bool h_changed, *d_changed;
  gpuErrchk(hipMalloc(&d_changed,sizeof(bool)));

  int zero = 0; // the distance zero from source


  uint64_t src;
  for(uint64_t j = 0; j < G.h_nnodes; ++j) {
    src = h_nodeId[j];
    if(h_nodeDegree[j] == 0) {
      std::cout << "Nodes with degree 0 start at: " << j << std::endl;
      std::cout << "Number of nodes with degree 0 : " << G.h_nnodes-1-j << std::endl;
      break;
    }
  if(h_level[src] == INF_NEW ) {

  gpuErrchk(hipMemcpy(&d_level[src],&zero, sizeof(zero), hipMemcpyHostToDevice));

  do {
    h_changed = false;
    gpuErrchk(hipMemcpy(d_changed, &h_changed, sizeof(h_changed), hipMemcpyHostToDevice));

    getLevel<<<numBlocks,blockSize>>>(G, G.h_nnodes, G.h_nedges, d_level, d_changed); // making it true all the time, so getting stuck in an infinite loop
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError() );

    gpuErrchk(hipMemcpy(&h_changed, d_changed, sizeof(h_changed), hipMemcpyDeviceToHost));
  } while(h_changed);

  gpuErrchk(hipMemcpy(h_level, d_level, G.h_nnodes*sizeof(int), hipMemcpyDeviceToHost));  

  }

}
  


 // assigning level 0 to nodes that have not been reached so far.. i.e., these are unreachable
  for(uint64_t s=0; s<G.h_nnodes; ++s) { 
  //  h_level[s] = (h_level[s] != INF_NEW) * h_level[s]; // this is optimal
     if(h_level[s] == INF_NEW) {
       h_level[s] = 0;
     }
  }

  cputimer.Stop();

  std::cout << "Time elapsed in assigning levels = " << cputimer.Elapsed() << " second" << std::endl;



#endif

// counting the number of nodes of each type
std::unordered_map<int,uint64_t> countPerLevel; // map of level:# nodes at that level
for(uint64_t i=0; i<G.h_nnodes; ++i) {
  countPerLevel[h_level[i]]++; 
}

int chunkSize = 32; // specifying the chunk size

std::cout << "Level : #nodes ; #holes" << std::endl;
unsigned holeSum = 0;
for(auto it=countPerLevel.begin(); it != countPerLevel.end(); ++it) {
  int  temp = chunkSize - ( (it->second) % chunkSize );
  std::cout << it->first << " : " << it->second << " ; " << temp  << std::endl;
  holeSum += temp;
}

std::cout << "total holes : " << holeSum << std::endl;




uint64_t * h_newId = (uint64_t*) malloc(sizeof(uint64_t)*G.h_nnodes); // stores the new id of the node, i.e., newId[i] = j means that new id assigned to nodes 'i' is 'j'.

// Step-1 : Assign the new id's to the nodes at level 0.

int maxLevel = 0;
uint64_t seqNum = 0; // the new id assigned to the nodes
for(uint64_t s = 0; s < G.h_nnodes; ++s) {
  if(h_level[s] == 0) {
    h_newId[s] = seqNum++;
  }
  else {
      maxLevel = max(h_level[s], maxLevel); // finding the number of levels in the bfs forest
    }
}

// Step-2 : Assign the new id's to the nodes at each level in a level-synchronous manner
seqNum = seqNum + ( chunkSize - (seqNum % chunkSize) ); // bump-up seqNum to the next multple of chunkSize

    // writing output to a file (for correctness check)

  const char filename[] = "bfs_output.txt";
  printf("Writing output to %s\n", filename);
  FILE *o = fopen(filename, "w");

  for(uint64_t i = 0; i < G.h_nnodes; i++) {
    fprintf(o, "%d: %d\n", i, h_level[i]);
  }

  fclose(o);

} // end of function



__global__ void getLevel(Graph G, uint64_t nnodes, uint64_t nedges, int* d_level, bool* d_changed) {
  uint64_t gid = threadIdx.x + blockDim.x * blockIdx.x;
  uint64_t src = gid; // node under consideration

  if(src >= nnodes)
    return;  // exit the kernel

  unsigned outDegree = G.getDegree(src);
  for(unsigned i=0; i<outDegree; ++i) {
      uint64_t dst = G. getDest(src,i); // get the i-th neighbor of src
      if(dst >= nnodes){
        return;
      }
//      unsigned wt = 1; // the edge weight is 1

  int altdist = d_level[src] + 1; // each edge has weight = 1
  if(altdist < d_level[dst]) { // a possible site for thread divergence
    int olddist = atomicMin(&d_level[dst], altdist);
    if(altdist < olddist) (*d_changed) =  true; // dist is updated to a lower value (another possible site for thread divergence)
  } 
}
}
