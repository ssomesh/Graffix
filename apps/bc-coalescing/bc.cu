#include "hip/hip_runtime.h"
#include "bc.h"

void bc_exact (Graph& G, double* nodeBC) {
  
  // declare the auxillary data structures for the bc computation
  
  unsigned n = G.nnodes;
  unsigned * d_sigma;
  int hops_from_source, * d_level, * d_hops_from_source;
  double * d_delta, * d_nodeBC;
  bool takeNextIter, * d_takeNextIter; // to decide whether to go for the next iteration or not
  gpuErrchk(hipMalloc(&d_sigma, sizeof(unsigned) * n));
  gpuErrchk(hipMalloc(&d_level, sizeof(int) * n));
  gpuErrchk(hipMalloc(&d_delta, sizeof(double) * n));
  gpuErrchk(hipMalloc(&d_nodeBC, sizeof(double) * n));

  gpuErrchk(hipMalloc(&d_hops_from_source, sizeof(int)));
  gpuErrchk(hipMalloc(&d_takeNextIter, sizeof(bool)));
  
  gpuErrchk(hipMemset(d_nodeBC, 0, sizeof(double)*n)); // initializing node bc

  
  unsigned blockSize = 512;
  unsigned gridSize = (n + blockSize - 1) / blockSize; // ceil(n/blockSize)
  
  for(unsigned s=0; s<n; ++s) {  // outer loop of Brandes' Algorithm
     hops_from_source = 0; // keeps track of the number of hops from source in the current iteration. 
     
     initialize<<<gridSize, blockSize>>>(d_sigma, d_delta, d_level, d_hops_from_source, s, n);
#ifdef DEBUG
     hipDeviceSynchronize();
     hipError_t errCode = hipPeekAtLastError();
     if (errCode != hipSuccess) {
       fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n",
       errCode, hipGetErrorString(errCode));
    }
#endif
    
    // forward pass 
    do {
      hipMemset(d_takeNextIter,false,sizeof(bool));
      bc_forward_pass<<<gridSize, blockSize>>>(G, d_sigma, d_level, d_hops_from_source, n, d_takeNextIter);
      hipDeviceSynchronize();
#ifdef DEBUG
     hipError_t errCode = hipPeekAtLastError();
     if (errCode != hipSuccess) {
       fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n",
       errCode, hipGetErrorString(errCode));
    }
#endif

      ++hops_from_source; // updating the level to process in the next iteration
//      gpuErrchk(hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(hops_from_source), hipMemcpyHostToDevice));
      incHop<<<1,1>>>(d_hops_from_source);

      gpuErrchk(hipMemcpy(&takeNextIter, d_takeNextIter, sizeof(bool), hipMemcpyDeviceToHost));
    }while(takeNextIter);
    
    
  // backward pass
  
  --hops_from_source;
  gpuErrchk(hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(hops_from_source), hipMemcpyHostToDevice));
  while(hops_from_source > 1) {
    bc_backward_pass<<<gridSize,blockSize>>>(G, d_sigma, d_delta, d_nodeBC, d_level, d_hops_from_source, n);
#ifdef DEBUG
     hipError_t errCode = hipPeekAtLastError();
     if (errCode != hipSuccess) {
       fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n",
       errCode, hipGetErrorString(errCode));
    }
#endif
    --hops_from_source;
    gpuErrchk(hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(hops_from_source), hipMemcpyHostToDevice));
  }

  accumulate_bc<<<gridSize, blockSize>>>(d_delta, d_nodeBC, d_level, s, n);
#ifdef DEBUG
     errCode = hipPeekAtLastError();
     if (errCode != hipSuccess) {
       fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n",
       errCode, hipGetErrorString(errCode));
    }
#endif
  hipDeviceSynchronize();



  } // end of outerloop of brandes' algorithm
   


    gpuErrchk(hipMemcpy(nodeBC, d_nodeBC, sizeof(double) * n, hipMemcpyDeviceToHost));

}



__global__ void initialize(unsigned* d_sigma, double* d_delta, int* d_level, int* d_hops_from_source, unsigned s, unsigned n) {
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= n) return;
    d_level[tid] = -1;
    d_delta[tid] = 0.0;
    d_sigma[tid] = 0;
    
    if(tid == s) { // for the source
      d_level[tid] = 0;
      d_sigma[tid] = 1;
      *d_hops_from_source = 0;
    }
  }

  __global__ void bc_forward_pass(Graph G, unsigned* d_sigma, int* d_level, int* d_hops_from_source, unsigned n, bool* d_takeNextIter) {
    unsigned u = blockIdx.x * blockDim.x + threadIdx.x;
    if(u >= n) return;

    // only processing the nodes at level '*d_hops_from_source' -- a level synchronous processing, though not work efficient
    if(d_level[u] == *d_hops_from_source) {  
       unsigned end = G.d_offset[u+1];
       for(unsigned i = G.d_offset[u]; i < end; ++i) { // going over the neighbors of u
          unsigned v = G.d_edges[i];
          if(d_level[v] == -1) {  // v is seen for the first time
            d_level[v] = *d_hops_from_source + 1; // no atomics required since this is benign data race due to level synchronous implementation
            *d_takeNextIter = true;
          }
          if(d_level[v] == *d_hops_from_source + 1) { // 'v' is indeed the neighbor of u
            atomicAdd(&d_sigma[v], d_sigma[u]);
          }
       }
    }
  }

  __global__ void bc_backward_pass(Graph G, unsigned* d_sigma, double* d_delta, double* d_nodeBC, int* d_level, int* d_hops_from_source, unsigned n) {
      
    unsigned u = blockIdx.x * blockDim.x + threadIdx.x;
    if(u >= n) return;

    if(d_level[u] == *d_hops_from_source - 1) { // backward traversal of DAG, one level at a time 
      
       unsigned end = G.d_offset[u+1];
       double sum = 0.0;
       for(unsigned i = G.d_offset[u]; i < end; ++i) { // going over the neighbors of u for which it is the predecessor in the DAG
          unsigned v = G.d_edges[i];
          if(d_level[v] == *d_hops_from_source) {
            sum += (1.0 * d_sigma[u]) / d_sigma[v] * (1.0 + d_delta[v]);
          }
       }

       d_delta[u] += sum;

    }

  }

  __global__ void accumulate_bc(double * d_delta, double* d_nodeBC, int* d_level, unsigned s, unsigned n) {
    
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= n || tid == s || d_level[tid] == -1) return;

    d_nodeBC[tid] += d_delta[tid]/2.0;

  }

__global__ void incHop(int* d_hops_from_source) {
    *d_hops_from_source = *d_hops_from_source + 1;
  }

__global__ void populateDegree(Graph G,int * d_nodeDegree, uint64_t* d_nodeId, uint64_t nnodes) {
  uint64_t gid = threadIdx.x + blockIdx.x * blockDim.x;
  if(gid >= nnodes) return;
  d_nodeDegree[gid] = G.getDegree(gid);
  d_nodeId[gid] = gid;
}

 void merge(int* a, int* b, uint64_t* c, uint64_t * c_aux, uint64_t lo, uint64_t mid, uint64_t hi, uint64_t n) {
       if (mid >= n) return;
       if (hi > n) hi = n;
       int i = lo, j = mid, ii = lo, jj = mid, k;
       for (k = lo; k < hi; k++) {
          if      (i == mid)       {  b[k] = a[j++];  c_aux[k]  = c[jj++]; } 
          else if (j == hi)        {  b[k] = a[i++];  c_aux[k]  = c[ii++]; }
          else if (a[j] > a[i])    {  b[k] = a[j++];  c_aux[k]  = c[jj++]; }// '>' means descending order
          else                     {  b[k] = a[i++];  c_aux[k]  = c[ii++]; }
       }
       // copy back
       for (k = lo; k < hi; k++) {
          a[k] = b[k];
          c[k] = c_aux[k];
       }
    }



void Merge_Sort_Par(int *a,int *b,uint64_t *c, uint64_t* c_aux, uint64_t n) //, int nThreads)
{
   omp_set_num_threads(16);
  uint64_t blockSize, start;
 
 for(blockSize=1;blockSize<n; blockSize=blockSize+blockSize){
  #pragma omp parallel for  private(start) schedule(static)
  for(start=0; start < n; start += blockSize + blockSize){
 // std::cout << "Get num threads " << omp_get_num_threads() << std::endl;
    merge(a, b, c, c_aux, start, start+blockSize, start + 2*blockSize, n);
}
 }

}




/*renumber and replicate the nodes */
void renumber_replicate(Graph& G) { 
  /* Step-1: renumber the nodes */

  // store the nodes' degrees in an array and sort the array in descending order
  int * h_nodeDegree = (int*) malloc(G.h_nnodes*sizeof(int));
  int * h_nodeDegree_aux = (int*) malloc(G.h_nnodes*sizeof(int)); // this is for the merge sort
  int * d_nodeDegree;
  uint64_t * h_nodeId = (uint64_t*) malloc(G.h_nnodes*sizeof(uint64_t));
  uint64_t * h_nodeId_aux = (uint64_t*) malloc(G.h_nnodes*sizeof(uint64_t));
  uint64_t * d_nodeId;
  gpuErrchk(hipMalloc(&d_nodeId,G.h_nnodes*sizeof(uint64_t))); 
  gpuErrchk(hipMalloc(&d_nodeDegree,G.h_nnodes*sizeof(int))); 
  unsigned blockSize = 256;
  unsigned numBlocks = (G.h_nnodes+blockSize-1)/blockSize;

  CPUTimer cputimer;
  cputimer.Start();

  populateDegree<<<numBlocks, blockSize>>>(G, d_nodeDegree,d_nodeId,G.h_nnodes);
  gpuErrchk(hipMemcpy(h_nodeDegree, d_nodeDegree, G.h_nnodes*sizeof(int), hipMemcpyDeviceToHost));  
  gpuErrchk(hipMemcpy(h_nodeId, d_nodeId, G.h_nnodes*sizeof(uint64_t), hipMemcpyDeviceToHost));  

  gpuErrchk(hipFree(d_nodeDegree));
  gpuErrchk(hipFree(d_nodeId));
  

  // sort the nodes in descending order and maintain another array to store the corresponding node id  
   Merge_Sort_Par(h_nodeDegree, h_nodeDegree_aux, h_nodeId, h_nodeId_aux, G.h_nnodes);

  cputimer.Stop();

  free(h_nodeDegree_aux);
  free(h_nodeId_aux);

  std::cout << "Time elapsed = " << cputimer.Elapsed() << " second" << std::endl;

  
#if 1

  int *h_level = (int*) malloc(G.h_nnodes*sizeof(int));
  int * d_level;
  gpuErrchk(hipMalloc(&d_level,G.h_nnodes*sizeof(int))); 

  cputimer.Start();

  levelInit<<<numBlocks,blockSize>>>(d_level,G.h_nnodes);

  gpuErrchk(hipMemcpy(h_level, d_level, G.h_nnodes*sizeof(int), hipMemcpyDeviceToHost));  // initializing h_level for the first iteration

  bool h_changed, *d_changed;
  gpuErrchk(hipMalloc(&d_changed,sizeof(bool)));

  int zero = 0; // the distance zero from source


  uint64_t src;
  for(uint64_t j = 0; j < G.h_nnodes; ++j) {
    src = h_nodeId[j];
    if(h_nodeDegree[j] == 0) {
      std::cout << "Nodes with degree 0 start at: " << j << std::endl;
      std::cout << "Number of nodes with degree 0 : " << G.h_nnodes-1-j << std::endl;
      break;
    }
  if(h_level[src] == INF_NEW ) {

  gpuErrchk(hipMemcpy(&d_level[src],&zero, sizeof(zero), hipMemcpyHostToDevice));

  do {
    h_changed = false;
    gpuErrchk(hipMemcpy(d_changed, &h_changed, sizeof(h_changed), hipMemcpyHostToDevice));

    getLevel<<<numBlocks,blockSize>>>(G, G.h_nnodes, G.h_nedges, d_level, d_changed); // making it true all the time, so getting stuck in an infinite loop
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError() );

    gpuErrchk(hipMemcpy(&h_changed, d_changed, sizeof(h_changed), hipMemcpyDeviceToHost));
  } while(h_changed);

  gpuErrchk(hipMemcpy(h_level, d_level, G.h_nnodes*sizeof(int), hipMemcpyDeviceToHost));  

  }

}
  


 // assigning level 0 to nodes that have not been reached so far.. i.e., these are unreachable
  for(uint64_t s=0; s<G.h_nnodes; ++s) { 
  //  h_level[s] = (h_level[s] != INF_NEW) * h_level[s]; // this is optimal
     if(h_level[s] == INF_NEW) {
       h_level[s] = 0;
     }
  }

  cputimer.Stop();

  std::cout << "Time elapsed in assigning levels = " << cputimer.Elapsed() << " second" << std::endl;



#endif

// counting the number of nodes of each type
std::unordered_map<int,uint64_t> countPerLevel; // map of level:# nodes at that level
for(uint64_t i=0; i<G.h_nnodes; ++i) {
  countPerLevel[h_level[i]]++; 
}

int chunkSize = 32; // specifying the chunk size

std::cout << "Level : #nodes ; #holes" << std::endl;
unsigned holeSum = 0;
for(auto it=countPerLevel.begin(); it != countPerLevel.end(); ++it) {
  int  temp = chunkSize - ( (it->second) % chunkSize );
  std::cout << it->first << " : " << it->second << " ; " << temp  << std::endl;
  holeSum += temp;
}

std::cout << "total holes : " << holeSum << std::endl;




uint64_t * h_newId = (uint64_t*) malloc(sizeof(uint64_t)*G.h_nnodes); // stores the new id of the node, i.e., newId[i] = j means that new id assigned to nodes 'i' is 'j'.

// Step-1 : Assign the new id's to the nodes at level 0.

int maxLevel = 0;
uint64_t seqNum = 0; // the new id assigned to the nodes
for(uint64_t s = 0; s < G.h_nnodes; ++s) {
  if(h_level[s] == 0) {
    h_newId[s] = seqNum++;
  }
  else {
      maxLevel = max(h_level[s], maxLevel); // finding the number of levels in the bfs forest
    }
}

// Step-2 : Assign the new id's to the nodes at each level in a level-synchronous manner
seqNum = seqNum + ( chunkSize - (seqNum % chunkSize) ); // bump-up seqNum to the next multple of chunkSize


    // writing output to a file (for correctness check)

  const char filename[] = "bfs_output.txt";
  printf("Writing output to %s\n", filename);
  FILE *o = fopen(filename, "w");

  for(uint64_t i = 0; i < G.h_nnodes; i++) {
    fprintf(o, "%d: %d\n", i, h_level[i]);
  }

  fclose(o);

} // end of function



__global__ void getLevel(Graph G, uint64_t nnodes, uint64_t nedges, int* d_level, bool* d_changed) {
  uint64_t gid = threadIdx.x + blockDim.x * blockIdx.x;
  uint64_t src = gid; // node under consideration

  if(src >= nnodes)
    return;  // exit the kernel

  unsigned outDegree = G.getDegree(src);
  for(unsigned i=0; i<outDegree; ++i) {
      uint64_t dst = G. getDest(src,i); // get the i-th neighbor of src
      if(dst >= nnodes){
        return;
      }
//      unsigned wt = 1; // the edge weight is 1

  int altdist = d_level[src] + 1; // each edge has weight = 1
  if(altdist < d_level[dst]) { // a possible site for thread divergence
    int olddist = atomicMin(&d_level[dst], altdist);
    if(altdist < olddist) (*d_changed) =  true; // dist is updated to a lower value (another possible site for thread divergence)
  } 
}
}
